#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <helper_cuda.h>

__global__ void vectorAdd(const float *A, const float *B, float *C,
                        int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < numElements) {
        C[idx] = A[idx] + B[idx] + 0.0f;
    }
}

int main() {
    hipError_t err = hipSuccess;

    int numElem = 50000;
    size_t sz = numElem * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElem);

    float *h_A = (float *) malloc(sz);
    float *h_B = (float *) malloc(sz);
    float *h_C = (float *) malloc(sz);
    
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for (int idx = 0; idx < numElem; idx++) {
        h_A[idx] = rand() / (float)RAND_MAX;
        h_B[idx] = rand() / (float)RAND_MAX;
    }

    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;

    err = hipMalloc((void **)&d_A, sz);
    err = hipMalloc((void **)&d_B, sz);
    err = hipMalloc((void **)&d_C, sz);

    hipMemcpy(d_A, h_A, sz, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sz, hipMemcpyHostToDevice);

    int thPerBlock = 256;
    int blksPerGrid = (numElem + thPerBlock - 1) / thPerBlock;
    printf("CUDA kernel launch with %d blocks of %d thread\n", blksPerGrid, thPerBlock);
    vectorAdd<<<blksPerGrid, thPerBlock>>>(d_A, d_B, d_C, numElem);
    
    err = hipMemcpy(h_C, d_C, sz, hipMemcpyDeviceToHost);

    // Verify that the result vector is correct
    for (int i = 0; i < numElem; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("\n\n\nDone..!\n\n\n");

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
}