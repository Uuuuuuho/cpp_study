#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

__global__ void inc_kernel(int *g_data, int inc_val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    g_data[idx] = g_data[idx] + inc_val;
}

bool correct_output(int *data, const int n, const int x) {
    for(int idx = 0; idx < n; idx++) {
        if(data[idx] != x) {
            printf("Error! data[%d] = %d, ref = %d\n", idx, data[idx], x);
            return false;
        }
    }
}

int main(int argc, char *argv[]) {
    int devId;
    hipDeviceProp_t deviceProps;

    printf("[%s] - starting...\n", argv[0]);

    devId = findCudaDevice(argc, (const char **)argv);

    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devId));
    printf("CUDA device [%s]\n", deviceProps.name);

    int n = 16 * 1024 * 1024;
    int nbytes = n * sizeof(int);
    int val = 26;

    int *a = 0;
    checkCudaErrors(hipHostMalloc((void **)&a, nbytes));
    memset(a, 0, nbytes);

    int *d_a = 0;
    checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
    checkCudaErrors(hipMemset(d_a, 255, nbytes));

    dim3 threads = dim3(512, 1);
    dim3 blocks = dim3(n / threads.x, 1);

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    checkCudaErrors(hipProfilerStart());
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);
    hipMemcpyAsync(d_a, a, nbytes, hipMemcpyHostToDevice, 0);
    inc_kernel<<<blocks, threads, 0, 0>>>(d_a, val);
    hipMemcpyAsync(a, d_a, nbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);
    checkCudaErrors(hipProfilerStop());

    unsigned long int cnt = 0;

    while(hipEventQuery(stop) == hipErrorNotReady) {
        cnt++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    printf("time spent executing by the GPU: %.f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.f\n", sdkGetTimerValue(&timer));
    printf("time spent executing by the GPU: %.f\n", gpu_time);

    bool bFinalResults = correct_output(a, n, val);

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipFree(d_a));

    exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}