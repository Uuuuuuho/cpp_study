#include "hip/hip_runtime.h"
#include "../common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv) {
    printf("%s starting...\n", argv[0]);

    int devCnt = 0;
    hipGetDeviceCount(&devCnt);

    if(devCnt == 0) {
        printf("There are no available devices that support CUDA\n");
    }
    else {
        printf("Detected %d CUDA capable devices\n", devCnt);
    }
    
    int dev = 0, driverVersion = 0, runtimeVersion = 0;
    CHECK(hipSetDevice(dev));
    hipDeviceProp_t devProp;
    CHECK(hipGetDeviceProperties(&devProp, dev));
    printf("Device %d: \%s\ \n", dev, devProp.name);

    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           devProp.major, devProp.minor);
    printf("  Total amount of global memory:                 %.2f GBytes (%llu "
           "bytes)\n", (float)devProp.totalGlobalMem / pow(1024.0, 3),
           (unsigned long long)devProp.totalGlobalMem);
    printf("  GPU Clock rate:                                %.0f MHz (%0.2f "
           "GHz)\n", devProp.clockRate * 1e-3f,
           devProp.clockRate * 1e-6f);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           devProp.memoryClockRate * 1e-3f);
    printf("  Memory Bus Width:                              %d-bit\n",
           devProp.memoryBusWidth);    

    if (devProp.l2CacheSize)
    {
        printf("  L2 Cache Size:                                 %d bytes\n",
               devProp.l2CacheSize);
    }

    printf("  Max Texture Dimension Size (x,y,z)             1D=(%d), "
           "2D=(%d,%d), 3D=(%d,%d,%d)\n", devProp.maxTexture1D,
           devProp.maxTexture2D[0], devProp.maxTexture2D[1],
           devProp.maxTexture3D[0], devProp.maxTexture3D[1],
           devProp.maxTexture3D[2]);
    printf("  Max Layered Texture Size (dim) x layers        1D=(%d) x %d, "
           "2D=(%d,%d) x %d\n", devProp.maxTexture1DLayered[0],
           devProp.maxTexture1DLayered[1], devProp.maxTexture2DLayered[0],
           devProp.maxTexture2DLayered[1],
           devProp.maxTexture2DLayered[2]);
    printf("  Total amount of constant memory:               %lu bytes\n",
           devProp.totalConstMem);
    printf("  Total amount of shared memory per block:       %lu bytes\n",
           devProp.sharedMemPerBlock);
    printf("  Total number of registers available per block: %d\n",
           devProp.regsPerBlock);
    printf("  Warp size:                                     %d\n",
           devProp.warpSize);
    printf("  Maximum number of threads per multiprocessor:  %d\n",
           devProp.maxThreadsPerMultiProcessor);
    printf("  Maximum number of threads per block:           %d\n",
           devProp.maxThreadsPerBlock);
    printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
           devProp.maxThreadsDim[0],
           devProp.maxThreadsDim[1],
           devProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
           devProp.maxGridSize[0],
           devProp.maxGridSize[1],
           devProp.maxGridSize[2]);
    printf("  Maximum memory pitch:                          %lu bytes\n",
           devProp.memPitch);

    exit(EXIT_SUCCESS);
}